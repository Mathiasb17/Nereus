#include "hip/hip_runtime.h"

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <hip/hip_vector_types.h>

#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>

#include "sph_kernel.cuh"
#include "sph_kernel_impl.cuh"

extern "C"
{
	void cudaInit(int argc, char **argv)
	{
		int devID;

		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		devID = findCudaDevice(argc, (const char **)argv);

		if (devID < 0)
		{
			printf("No CUDA Capable devices found, exiting...\n");
			exit(EXIT_SUCCESS);
		}
	}

	void cudaGLInit(int argc, char **argv)
	{
		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		findCudaGLDevice(argc, (const char **)argv);
	}

	void allocateArray(void **devPtr, size_t size)
	{
		checkCudaErrors(hipMalloc(devPtr, size));
	}

	void freeArray(void *devPtr)
	{
		checkCudaErrors(hipFree(devPtr));
	}

	void threadSync()
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

	void copyArrayToDevice(void *device, const void *host, int offset, int size)
	{
		checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

	void registerGLBufferObject(uint vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
					cudaGraphicsMapFlagsNone));
	}

	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
	}

	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
					*cuda_vbo_resource));
		return ptr;
	}

	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

	void copyArrayFromDevice(void *host, const void *device,
			struct hipGraphicsResource **cuda_vbo_resource, int size)
	{
		if (cuda_vbo_resource)
		{
			device = mapGLBufferObject(cuda_vbo_resource);
		}

		checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

		if (cuda_vbo_resource)
		{
			unmapGLBufferObject(*cuda_vbo_resource);
		}
	}

	void setParameters(SphSimParams *hostParams)
	{
		// copy parameters to constant memory
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sph_params), hostParams, sizeof(SphSimParams)));
	}

	//Round a / b to nearest higher integer value
	uint iDivUp(uint a, uint b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

	// compute grid and thread block size for a given number of elements
	void computeGridSize(uint n, uint blockSize, uint &numBlocks, uint &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

	void integrateSystem(float *pos,
			float *vel,
			float deltaTime,
			uint numParticles)
	{
		thrust::device_ptr<float4> d_pos4((float4 *)pos);
		thrust::device_ptr<float4> d_vel4((float4 *)vel);

		thrust::for_each(
				thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
				thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles)),
				integrate_functor(deltaTime));
	}

	void calcHash(uint  *gridParticleHash,
			uint  *gridParticleIndex,
			float *pos,
			int    numParticles)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		// execute the kernel
		calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed");
	}

	void reorderDataAndFindCellStart(uint  *cellStart,
			uint  *cellEnd,
			float *sortedPos,
			float *sortedVel,
			float *sortedDens,
			float *sortedPres,
			float *sortedForces,
			float *sortedCol,
			uint  *gridParticleHash,
			uint  *gridParticleIndex,
			float *oldPos,
			float *oldVel,
			float *oldDens,
			float *oldPres,
			float *oldForces,
			float *oldCol,
			uint   numParticles,
			uint   numCells)
	{
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		// set all cells to empty
		checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));

#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldDensTex, oldDens, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldPresTex, oldPres, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldForcesTex, oldForces, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldColTex, oldCol, numParticles*sizeof(float4)));
#endif

		uint smemSize = sizeof(uint)*(numThreads+1);
		reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
				cellStart,
				cellEnd,
				(float4 *) sortedPos,
				(float4 *) sortedVel,
				(float *) sortedDens,
				(float *) sortedPres,
				(float4 *) sortedForces,
				(float4 *) sortedCol,
				gridParticleHash,
				gridParticleIndex,
				(float4 *) oldPos,
				(float4 *) oldVel,
				(float *) oldDens,
				(float *) oldPres,
				(float4 *) oldForces,
				(float4 *) oldCol,
				numParticles);
		getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
		checkCudaErrors(hipUnbindTexture(oldPosTex));
		checkCudaErrors(hipUnbindTexture(oldVelTex));
		checkCudaErrors(hipUnbindTexture(oldDensTex));
		checkCudaErrors(hipUnbindTexture(oldPresTex));
		checkCudaErrors(hipUnbindTexture(oldForcesTex));
		checkCudaErrors(hipUnbindTexture(oldColTex));
#endif
	}

	void computeDensityPressure(float *newDens, float* newPres,
			float *sortedPos,
			float *sortedVel,
			float *sortedDens,
			float *sortedPres,
			float *sortedForces,
			float *sortedCol,
			uint  *gridParticleIndex,
			uint  *cellStart,
			uint  *cellEnd,
			uint   numParticles,
			uint   numCells)
	{
#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldDensTex, sortedVel, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldPresTex, sortedVel, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldForcesTex, sortedVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldColTex, sortedVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
		checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
#endif

		// thread per particle
		uint numThreads, numBlocks;
		computeGridSize(numParticles, 64, numBlocks, numThreads);

		// execute the kernel
		
		computeDensityPressure<<< numBlocks, numThreads >>>((float*)newDens, (float*)newPres,
				(float4 *)sortedPos,
				(float4 *)sortedVel,
				(float *)sortedDens,
				(float *)sortedPres,
				(float4 *)sortedForces,
				(float4 *)sortedCol,
				gridParticleIndex,
				cellStart,
				cellEnd,
				numParticles);

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed");

#if USE_TEX
		checkCudaErrors(hipUnbindTexture(oldPosTex));
		checkCudaErrors(hipUnbindTexture(oldVelTex));
		checkCudaErrors(hipUnbindTexture(oldDensTex));
		checkCudaErrors(hipUnbindTexture(oldPresTex));
		checkCudaErrors(hipUnbindTexture(oldForcesTex));
		checkCudaErrors(hipUnbindTexture(oldColTex));
		checkCudaErrors(hipUnbindTexture(cellStartTex));
		checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
	}


	void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex, uint numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<uint>(dGridParticleHash),
				thrust::device_ptr<uint>(dGridParticleHash + numParticles),
				thrust::device_ptr<uint>(dGridParticleIndex));
	}

}
