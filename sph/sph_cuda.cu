#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <hip/hip_vector_types.h>

#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "sph_kernel.cuh"
#include "sph_kernel_impl.cuh"

extern "C"
{

	/********************************
	*  SORT AND THRUST REDUCTIONS  *
	********************************/

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	float maxDensity(float* dDensities, unsigned int numParticles)
	{
		
		float res = *thrust::max_element(thrust::device, 
				thrust::device_ptr<float>(dDensities),
				thrust::device_ptr<float>(dDensities+numParticles)
				);

		return res;
	}
	
//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void sortParticles(unsigned int *dGridParticleHash, unsigned int *dGridParticleIndex, unsigned int numParticles)
	{
		thrust::sort_by_key(thrust::device_ptr<unsigned int>(dGridParticleHash),
				thrust::device_ptr<unsigned int>(dGridParticleHash + numParticles),
				thrust::device_ptr<unsigned int>(dGridParticleIndex));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void cudaInit(int argc, char **argv)
	{
		int devID;

		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		devID = findCudaDevice(argc, (const char **)argv);

		if (devID < 0)
		{
			printf("No CUDA Capable devices found, exiting...\n");
			exit(EXIT_SUCCESS);
		}
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void cudaGLInit(int argc, char **argv)
	{
		// use command-line specified CUDA device, otherwise use device with highest Gflops/s
		findCudaGLDevice(argc, (const char **)argv);
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void allocateArray(void **devPtr, size_t size)
	{
		checkCudaErrors(hipMalloc(devPtr, size));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void freeArray(void *devPtr)
	{
		checkCudaErrors(hipFree(devPtr));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void threadSync()
	{
		checkCudaErrors(hipDeviceSynchronize());
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void copyArrayToDevice(void *device, const void *host, int offset, int size)
	{
		checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void registerGLBufferObject(unsigned int vbo, struct hipGraphicsResource **cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
					cudaGraphicsMapFlagsNone));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
	{
		void *ptr;
		checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
					*cuda_vbo_resource));
		return ptr;
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
	{
		checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void copyArrayFromDevice(void *host, const void *device,
			struct hipGraphicsResource **cuda_vbo_resource, int size)
	{
		if (cuda_vbo_resource)
		{
			device = mapGLBufferObject(cuda_vbo_resource);
		}

		checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

		if (cuda_vbo_resource)
		{
			unmapGLBufferObject(*cuda_vbo_resource);
		}
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void setParameters(SphSimParams *hostParams)
	{
		// copy parameters to constant memory
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sph_params), hostParams, sizeof(SphSimParams)));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	//Round a / b to nearest higher integer value
	unsigned int iDivUp(unsigned int a, unsigned int b)
	{
		return (a % b != 0) ? (a / b + 1) : (a / b);
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	// compute grid and thread block size for a given number of elements
	void computeGridSize(unsigned int n, unsigned int blockSize, unsigned int &numBlocks, unsigned int &numThreads)
	{
		numThreads = min(blockSize, n);
		numBlocks = iDivUp(n, numThreads);
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void integrateSystem(float *pos,
			float *vel,
			float *forces,
			float deltaTime,
			unsigned int numParticles)
	{
		thrust::device_ptr<float4> d_pos4((float4 *)pos);
		thrust::device_ptr<float4> d_vel4((float4 *)vel);
		thrust::device_ptr<float4> d_forces4((float4 *)forces);

		thrust::for_each(
				thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4, d_forces4)),
				thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles, d_forces4+numParticles)),
				integrate_functor(deltaTime));
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	void calcHash(unsigned int  *gridParticleHash,
			unsigned int  *gridParticleIndex,
			float *pos,
			int    numParticles)
	{
		unsigned int numThreads, numBlocks;
		computeGridSize(numParticles, 256, numBlocks, numThreads);

		// execute the kernel
		calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
				gridParticleIndex,
				(float4 *) pos,
				numParticles);

		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed");
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 

	/*********************
	*  REORDERING CALL  *
	*********************/

	void reorderDataAndFindCellStartDBoundary(unsigned int *cellStart,
											unsigned int *cellEnd,
											float *sortedPos,
											float *sortedVbi,
											unsigned int *gridParticleHash,
											unsigned int *gridParticleIndex,
											float *oldPos,
											float *oldVbi,
											unsigned int numBoundaries,
											unsigned int numCells
											)
	{
		unsigned int numThreads, numBlocks;
		computeGridSize(numBoundaries, 64, numBlocks, numThreads);

		checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(unsigned int)));

#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldBoundaryPosTex, oldPos, numBoundaries*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldBoundaryVbiTex, oldVbi, numBoundaries*sizeof(float)));

#endif
		unsigned int smemSize = sizeof(unsigned int)*(numThreads+1);

		reorderDataAndFindCellStartDBoundary<<<numBlocks, numThreads, smemSize>>>(
				cellStart,
				cellEnd,
				(float4*) sortedPos,
				(float*)  sortedVbi,
				(unsigned int *)gridParticleHash,
				(unsigned int *)gridParticleIndex,
				(float4*) oldPos,
				(float*)  oldVbi,
				numBoundaries);

		getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");


#if USE_TEX
		checkCudaErrors(hipBindTexture(0, cellBoundaryStartTex, cellStart, numCells*sizeof(unsigned int)));
		checkCudaErrors(hipBindTexture(0, cellBoundaryEndTex, cellEnd, numCells*sizeof(unsigned int)));
#endif
	}

	void reorderDataAndFindCellStart(unsigned int  *cellStart,
			unsigned int  *cellEnd,
			float *sortedPos,
			float *sortedVel,
			float *sortedDens,
			float *sortedPres,
			float *sortedForces,
			float *sortedCol,
			unsigned int  *gridParticleHash,
			unsigned int  *gridParticleIndex,
			float *oldPos,
			float *oldVel,
			float *oldDens,
			float *oldPres,
			float *oldForces,
			float *oldCol,
			unsigned int   numParticles,
			unsigned int   numCells)
	{
		unsigned int numThreads, numBlocks;
		computeGridSize(numParticles, 64, numBlocks, numThreads);

		// set all cells to empty
		checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(unsigned int)));

#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldDensTex, oldDens, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldPresTex, oldPres, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldForcesTex, oldForces, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldColTex, oldCol, numParticles*sizeof(float4)));
#endif

		unsigned int smemSize = sizeof(unsigned int)*(numThreads+1);
		reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
				cellStart,
				cellEnd,
				(float4 *) sortedPos,
				(float4 *) sortedVel,
				NULL,
				NULL,
				NULL,
				NULL,
				gridParticleHash,
				gridParticleIndex,
				(float4 *) oldPos,
				(float4 *) oldVel,
				NULL,
				NULL,
				NULL,
				NULL,
				numParticles);
		getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
		checkCudaErrors(hipUnbindTexture(oldPosTex));
		checkCudaErrors(hipUnbindTexture(oldVelTex));
		checkCudaErrors(hipUnbindTexture(oldDensTex));
		checkCudaErrors(hipUnbindTexture(oldPresTex));
		checkCudaErrors(hipUnbindTexture(oldForcesTex));
		checkCudaErrors(hipUnbindTexture(oldColTex));
#endif
	}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	/******************************
	*  SPH COMPUTATION WITH EOS  *
	******************************/
	void computeDensityPressure(
			float *sortedPos,
			float *sortedVel,
			float *sortedDens,
			float *sortedPres,
			float *sortedForces,
			float *sortedCol,
			float *sortedBoundaryPos,
			float *sortedBoundaryVbi,
			unsigned int  *gridParticleIndex,
			unsigned int  *cellStart,
			unsigned int  *cellEnd,
			unsigned int *gridBoundaryIndex,
			unsigned int *cellBoundaryStart,
			unsigned int *cellBoundaryEnd,
			unsigned int   numParticles,
			unsigned int   numCells,
			unsigned int   numBoundaries)
	{
#if USE_TEX
		checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldDensTex, sortedDens, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldPresTex, sortedPres, numParticles*sizeof(float)));
		checkCudaErrors(hipBindTexture(0, oldForcesTex, sortedForces, numParticles*sizeof(float4)));
		checkCudaErrors(hipBindTexture(0, oldColTex, sortedCol, numParticles*sizeof(float4)));

		checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(unsigned int)));
		checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(unsigned int)));

		/*checkCudaErrors(hipBindTexture(0, oldBoundaryPosTex, sortedBoundaryPos, numBoundaries*sizeof(float4)));*/
		/*checkCudaErrors(hipBindTexture(0, oldBoundaryVbiTex, sortedBoundaryVbi, numBoundaries*sizeof(float)));*/

		//TODO missing cudamalloc in updateBoundary
		/*checkCudaErrors(hipBindTexture(0, cellBoundaryStartTex, cellBoundaryStart, numCells*sizeof(unsigned int)));*/
		/*checkCudaErrors(hipBindTexture(0, cellBoundaryEndTex, cellBoundaryEnd, numCells*sizeof(unsigned int)));*/
#endif

		// thread per particle
		unsigned int numThreads, numBlocks;
		computeGridSize(numParticles, 64, numBlocks, numThreads);

		// execute the kernel
		computeDensityPressure<<<numBlocks, numThreads>>>(
				(float4 *)sortedPos,
				(float4 *)sortedVel,
				(float  *)sortedDens,
				(float  *)sortedPres,
				(float4 *)sortedForces,
				(float4 *)sortedCol,
				(float4 *)sortedBoundaryPos,
				(float  *)sortedBoundaryVbi,
				gridParticleIndex,    // input: sorted particle indices
				cellStart,
				cellEnd,
				gridBoundaryIndex,
				cellBoundaryStart,
				cellBoundaryEnd,
				numParticles
		);
		
		hipDeviceSynchronize();

		computeForces<<< numBlocks, numThreads >>>(
			  (float4*) sortedPos,               // input: sorted positions
			  (float4*) sortedVel,               // input: sorted velocities
			  (float*) sortedDens,               // input: sorted velocities
			  (float*) sortedPres,               // input: sorted velocities
			  (float4*) sortedForces,            // input: sorted velocities
			  (float4*) sortedCol,               // input: sorted velocities
			  gridParticleIndex,    // input: sorted particle indices
			  cellStart,
			  cellEnd,
			  numParticles);


		// check if kernel invocation generated an error
		getLastCudaError("Kernel execution failed");

#if USE_TEX
		checkCudaErrors(hipUnbindTexture(oldPosTex));
		checkCudaErrors(hipUnbindTexture(oldVelTex));
		checkCudaErrors(hipUnbindTexture(oldDensTex));
		checkCudaErrors(hipUnbindTexture(oldPresTex));
		checkCudaErrors(hipUnbindTexture(oldForcesTex));
		checkCudaErrors(hipUnbindTexture(oldColTex));
		checkCudaErrors(hipUnbindTexture(cellStartTex));
		checkCudaErrors(hipUnbindTexture(cellEndTex));

		/*checkCudaErrors(hipUnbindTexture(oldBoundaryPosTex));*/
		/*checkCudaErrors(hipUnbindTexture(oldBoundaryVbiTex));*/

		//TODO
		/*checkCudaErrors(hipUnbindTexture(cellBoundaryStartTex));*/
		/*checkCudaErrors(hipUnbindTexture(cellBoundaryEndTex));*/
#endif
	}
}
