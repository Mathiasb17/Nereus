#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>
#include <helper_cuda_gl.h>
#include <hip/hip_vector_types.h>

#include <helper_functions.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

#include "sph_kernel.cuh"
#include "sph_kernel_impl.cuh"

EXTERN_C_BEGIN

	/********************************
	*  SORT AND THRUST REDUCTIONS  *
	********************************/

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
SReal maxDensity(SReal* dDensities, SUint numParticles)
{
	
	SReal res = *thrust::max_element(thrust::device, 
			thrust::device_ptr<SReal>(dDensities),
			thrust::device_ptr<SReal>(dDensities+numParticles)
			);

	return res;
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
SVec3 maxVelocity(SReal* dVelocities, SUint numParticles)
{
	SVec3 res = *thrust::max_element(thrust::device,
			thrust::device_ptr<SVec3>((SVec3*)dVelocities),
			thrust::device_ptr<SVec3>((SVec3*)dVelocities+numParticles),
			comp());
	return res;
}
	
//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void sortParticles(SUint *dGridParticleHash, SUint *dGridParticleIndex, SUint numParticles)
{
	thrust::sort_by_key(thrust::device_ptr<SUint>(dGridParticleHash),
			thrust::device_ptr<SUint>(dGridParticleHash + numParticles),
			thrust::device_ptr<SUint>(dGridParticleIndex));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void cudaInit(int argc, char **argv)
{
	int devID;

	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	devID = findCudaDevice(argc, (const char **)argv);

	if (devID < 0)
	{
		printf("No CUDA Capable devices found, exiting...\n");
		exit(EXIT_SUCCESS);
	}
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void cudaGLInit(int argc, char **argv)
{
	// use command-line specified CUDA device, otherwise use device with highest Gflops/s
	findCudaGLDevice(argc, (const char **)argv);
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void allocateArray(void **devPtr, size_t size)
{
	checkCudaErrors(hipMalloc(devPtr, size));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void freeArray(void *devPtr)
{
	checkCudaErrors(hipFree(devPtr));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void threadSync()
{
	checkCudaErrors(hipDeviceSynchronize());
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void copyArrayToDevice(void *device, const void *host, int offset, int size)
{
	checkCudaErrors(hipMemcpy((char *) device + offset, host, size, hipMemcpyHostToDevice));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void registerGLBufferObject(SUint vbo, struct hipGraphicsResource **cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
				cudaGraphicsMapFlagsNone));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource)
{
	void *ptr;
	checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
	size_t num_bytes;
	checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&ptr, &num_bytes,
				*cuda_vbo_resource));
	return ptr;
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource)
{
	checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void copyArrayFromDevice(void *host, const void *device,
		struct hipGraphicsResource **cuda_vbo_resource, int size)
{
	if (cuda_vbo_resource)
	{
		device = mapGLBufferObject(cuda_vbo_resource);
	}

	checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

	if (cuda_vbo_resource)
	{
		unmapGLBufferObject(*cuda_vbo_resource);
	}
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void setParameters(SphSimParams *hostParams)
{
	// copy parameters to constant memory
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(sph_params), hostParams, sizeof(SphSimParams)));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
//Round a / b to nearest higher integer value
SUint iDivUp(SUint a, SUint b)
{
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
// compute grid and thread block size for a given number of elements
void computeGridSize(SUint n, SUint blockSize, SUint &numBlocks, SUint &numThreads)
{
	numThreads = min(blockSize, n);
	numBlocks = iDivUp(n, numThreads);
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void integrateSystem(SReal *pos,
		SReal *vel,
		SReal *forces,
		SReal deltaTime,
		SUint numParticles)
{
	thrust::device_ptr<SVec4> d_pos4((SVec4 *)pos);
	thrust::device_ptr<SVec4> d_vel4((SVec4 *)vel);
	thrust::device_ptr<SVec4> d_forces4((SVec4 *)forces);

	thrust::for_each(
			thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4, d_forces4)),
			thrust::make_zip_iterator(thrust::make_tuple(d_pos4+numParticles, d_vel4+numParticles, d_forces4+numParticles)),
			integrate_functor(deltaTime));
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void calcHash(SUint  *gridParticleHash,
		SUint  *gridParticleIndex,
		SReal *pos,
		int    numParticles)
{
	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 256, numBlocks, numThreads);

	// execute the kernel
	calcHashD<<< numBlocks, numThreads >>>(gridParticleHash,
			gridParticleIndex,
			(SVec4 *) pos,
			numParticles);

	// check if kernel invocation generated an error
	getLastCudaError("calcHash Failed");
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
/*********************
 *  REORDERING CALL  *
 *********************/
void reorderDataAndFindCellStartDBoundary(SUint *cellStart,
										SUint *cellEnd,
										SReal *sortedPos,
										SReal *sortedVbi,
										SUint *gridParticleHash,
										SUint *gridParticleIndex,
										SReal *oldPos,
										SReal *oldVbi,
										SUint numBoundaries,
										SUint numCells
										)
{
	SUint numThreads, numBlocks;
	computeGridSize(numBoundaries, 64, numBlocks, numThreads);

	checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(SUint)));

#if USE_TEX
	checkCudaErrors(hipBindTexture(0, oldBoundaryPosTex, oldPos, numBoundaries*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldBoundaryVbiTex, oldVbi, numBoundaries*sizeof(SReal)));
#endif
	SUint smemSize = sizeof(SUint)*(numThreads+1);

	reorderDataAndFindCellStartDBoundary<<<numBlocks, numThreads, smemSize>>>(
			cellStart,
			cellEnd,
			(SVec4*) sortedPos,
			(SReal*)  sortedVbi,
			(SUint *)gridParticleHash,
			(SUint *)gridParticleIndex,
			(SVec4*) oldPos,
			(SReal*)  oldVbi,
			numBoundaries);

	getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");
#if USE_TEX
	checkCudaErrors(hipBindTexture(0, cellBoundaryStartTex, cellStart, numCells*sizeof(SUint)));
	checkCudaErrors(hipBindTexture(0, cellBoundaryEndTex, cellEnd, numCells*sizeof(SUint)));
#endif
}

void reorderDataAndFindCellStart(SUint  *cellStart,
		SUint  *cellEnd,
		SReal *sortedPos,
		SReal *sortedVel,
		SReal *sortedDens,
		SReal *sortedPres,
		SReal *sortedForces,
		SReal *sortedCol,
		SUint  *gridParticleHash,
		SUint  *gridParticleIndex,
		SReal *oldPos,
		SReal *oldVel,
		SReal *oldDens,
		SReal *oldPres,
		SReal *oldForces,
		SReal *oldCol,
		SUint   numParticles,
		SUint   numCells)
{
	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);

	// set all cells to empty
	checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells*sizeof(SUint)));

#if USE_TEX
	checkCudaErrors(hipBindTexture(0, oldPosTex, oldPos, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldVelTex, oldVel, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDensTex, oldDens, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPresTex, oldPres, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldForcesTex, oldForces, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldColTex, oldCol, numParticles*sizeof(SVec4)));
#endif

	SUint smemSize = sizeof(SUint)*(numThreads+1);
	reorderDataAndFindCellStartD<<< numBlocks, numThreads, smemSize>>>(
			cellStart,
			cellEnd,
			(SVec4 *) sortedPos,
			(SVec4 *) sortedVel,
			NULL,
			sortedPres,
			NULL,
			NULL,
			gridParticleHash,
			gridParticleIndex,
			(SVec4 *) oldPos,
			(SVec4 *) oldVel,
			NULL,
			oldPres,
			NULL,
			NULL,
			numParticles);
	getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");

#if USE_TEX
	checkCudaErrors(hipUnbindTexture(oldPosTex));
	checkCudaErrors(hipUnbindTexture(oldVelTex));
	checkCudaErrors(hipUnbindTexture(oldDensTex));
	checkCudaErrors(hipUnbindTexture(oldPresTex));
	checkCudaErrors(hipUnbindTexture(oldForcesTex));
	checkCudaErrors(hipUnbindTexture(oldColTex));
#endif
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
	/******************************
	*  SPH COMPUTATION WITH EOS  *
	******************************/
void computeDensityPressure(
		SReal *sortedPos,
		SReal *sortedVel,
		SReal *sortedDens,
		SReal *sortedPres,
		SReal *sortedForces,
		SReal *sortedCol,
		SReal *sortedBoundaryPos,
		SReal *sortedBoundaryVbi,
		SUint  *gridParticleIndex,
		SUint  *cellStart,
		SUint  *cellEnd,
		SUint *gridBoundaryIndex,
		SUint *cellBoundaryStart,
		SUint *cellBoundaryEnd,
		SUint   numParticles,
		SUint   numCells,
		SUint   numBoundaries)
{
#if USE_TEX
	checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDensTex, sortedDens, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPresTex, sortedPres, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldForcesTex, sortedForces, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldColTex, sortedCol, numParticles*sizeof(SVec4)));

	checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(SUint)));
	checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(SUint)));
#endif

	// thread per particle
	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);

	// execute the kernel
	computeDensityPressure<<<numBlocks, numThreads>>>(
			(SVec4 *)sortedPos,
			(SVec4 *)sortedVel,
			(SReal  *)sortedDens,
			(SReal  *)sortedPres,
			(SVec4 *)sortedForces,
			(SVec4 *)sortedCol,
			(SVec4 *)sortedBoundaryPos,
			(SReal  *)sortedBoundaryVbi,
			gridParticleIndex,    // input: sorted particle indices
			cellStart,
			cellEnd,
			gridBoundaryIndex,
			cellBoundaryStart,
			cellBoundaryEnd,
			numParticles
	);

	/*SReal maxd =  maxDensity(sortedDens, numParticles);*/
	/*printf("maxd = %f\n", maxd);*/
	
	/*hipDeviceSynchronize();*/

	computeForces<<< numBlocks, numThreads >>>(
		  (SVec4*) sortedPos,               // input: sorted positions
		  (SVec4*) sortedVel,               // input: sorted velocities
		  (SReal*) sortedDens,               // input: sorted velocities
		  (SReal*) sortedPres,               // input: sorted velocities
		  (SVec4*) sortedForces,            // input: sorted velocities
		  (SVec4*) sortedCol,               // input: sorted velocities
		  gridBoundaryIndex,
		  (SVec4*) sortedBoundaryPos,
		  (SReal*) sortedBoundaryVbi,
		  gridParticleIndex,    // input: sorted particle indices
		  cellStart,
		  cellEnd,
		  cellBoundaryStart,
		  cellBoundaryEnd,
		  numParticles);


	// check if kernel invocation generated an error
	getLastCudaError("Kernel execution failed");

#if USE_TEX
	checkCudaErrors(hipUnbindTexture(oldPosTex));
	checkCudaErrors(hipUnbindTexture(oldVelTex));
	checkCudaErrors(hipUnbindTexture(oldDensTex));
	checkCudaErrors(hipUnbindTexture(oldPresTex));
	checkCudaErrors(hipUnbindTexture(oldForcesTex));
	checkCudaErrors(hipUnbindTexture(oldColTex));
	checkCudaErrors(hipUnbindTexture(cellStartTex));
	checkCudaErrors(hipUnbindTexture(cellEndTex));
#endif
}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
/***********
*  IISPH  *
***********/
void predictAdvection(SReal* sortedPos,
	SReal                       * sortedVel,
	SReal                       * sortedDens,
	SReal                       * sortedPres,
	SReal                       * sortedForces,
	SReal                       * sortedCol,
	SUint                * cellStart,
	SUint                * cellEnd,
	SUint                * gridParticleIndex,
	SReal						* sortedBoundaryPos,
	SReal						* sortedBoundaryVbi,
	SUint                * cellBoundaryStart,
	SUint                * cellBoundaryEnd,
	SUint                * gridBoundaryIndex,
	SReal                       * sortedDensAdv,
	SReal                       * sortedDensCorr,
	SReal                       * sortedP_l,
	SReal                       * sortedPreviousP,
	SReal                       * sortedAii,
	SReal                       * sortedVelAdv,
	SReal                       * sortedForcesAdv,
	SReal                       * sortedForcesP,
	SReal                       * sortedDiiFluid,
	SReal                       * sortedDiiBoundary,
	SReal                       * sortedSumDij,
	SReal                       * sortedNormal,
	SUint numParticles,
	SUint numBoundaries,
	SUint numCells)
{
#if USE_TEX
	checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDensTex, sortedDens, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPresTex, sortedPres, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldForcesTex, sortedForces, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldColTex, sortedCol, numParticles*sizeof(SVec4)));

	checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(SUint)));
	checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(SUint)));

	checkCudaErrors(hipBindTexture(0, oldDensAdvTex, sortedDensAdv, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldDensCorrTex, sortedDensCorr, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldP_lTex, sortedP_l, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPreviousPTex, sortedPreviousP, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldAiiTex, sortedAii, numParticles*sizeof(SReal)));

	checkCudaErrors(hipBindTexture(0, oldVelAdvTex, sortedVelAdv, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldForcesAdvTex, sortedForcesAdv, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldForcesPTex, sortedForcesP, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDiiFluidTex, sortedDiiFluid, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDiiBoundaryTex, sortedDiiBoundary, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldSumDijTex, sortedSumDij, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldNormalTex, sortedNormal, numParticles*sizeof(SVec4)));
#endif

	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);

	computeIisphDensity<<<numBlocks, numThreads>>>(
			(SVec4*) sortedPos,
			(SVec4*) sortedVel,
			sortedDens,
			sortedPres,
			(SVec4*) sortedForces,
			(SVec4*) sortedCol,
			cellStart,
			cellEnd,
			gridParticleIndex,
			(SVec4*) sortedBoundaryPos,
			sortedBoundaryVbi,
			cellBoundaryStart,
			cellBoundaryEnd,
			gridBoundaryIndex,
			sortedDensAdv,
			sortedDensCorr,
			sortedP_l,
			sortedPreviousP,
			sortedAii,
			(SVec4*) sortedVelAdv,
			(SVec4*) sortedForcesAdv,
			(SVec4*) sortedForcesP,
			(SVec4*) sortedDiiFluid,
			(SVec4*) sortedDiiBoundary,
			(SVec4*) sortedSumDij,
			(SVec4*) sortedNormal,
			numParticles,
			numBoundaries,
			numCells);


	/*hipDeviceSynchronize();*/

	computeDisplacementFactor<<<numBlocks, numThreads>>>(
			(SVec4*) sortedPos,
			(SVec4*) sortedVel,
			sortedDens,
			sortedPres,
			(SVec4*) sortedForces,
			(SVec4*) sortedCol,
			cellStart,
			cellEnd,
			gridParticleIndex,
			(SVec4*) sortedBoundaryPos,
			sortedBoundaryVbi,
			cellBoundaryStart,
			cellBoundaryEnd,
			gridBoundaryIndex,
			sortedDensAdv,
			sortedDensCorr,
			sortedP_l,
			sortedPreviousP,
			sortedAii,
			(SVec4*) sortedVelAdv,
			(SVec4*) sortedForcesAdv,
			(SVec4*) sortedForcesP,
			(SVec4*) sortedDiiFluid,
			(SVec4*) sortedDiiBoundary,
			(SVec4*) sortedSumDij,
			(SVec4*) sortedNormal,
			numParticles,
			numBoundaries,
			numCells);

	/*hipDeviceSynchronize();*/

	computeAdvectionFactor<<<numBlocks, numThreads>>>(
			(SVec4*) sortedPos,
			(SVec4*) sortedVel,
			sortedDens,
			sortedPres,
			(SVec4*) sortedForces,
			(SVec4*) sortedCol,
			cellStart,
			cellEnd,
			gridParticleIndex,
			(SVec4*) sortedBoundaryPos,
			sortedBoundaryVbi,
			cellBoundaryStart,
			cellBoundaryEnd,
			gridBoundaryIndex,
			sortedDensAdv,
			sortedDensCorr,
			sortedP_l,
			sortedPreviousP,
			sortedAii,
			(SVec4*) sortedVelAdv,
			(SVec4*) sortedForcesAdv,
			(SVec4*) sortedForcesP,
			(SVec4*) sortedDiiFluid,
			(SVec4*) sortedDiiBoundary,
			(SVec4*) sortedSumDij,
			(SVec4*) sortedNormal,
			numParticles,
			numBoundaries,
			numCells);

	/*hipDeviceSynchronize();*/
#if USE_TEX
	checkCudaErrors(hipUnbindTexture(oldPosTex));
	checkCudaErrors(hipUnbindTexture(oldVelTex));
	checkCudaErrors(hipUnbindTexture(oldDensTex));
	checkCudaErrors(hipUnbindTexture(oldPresTex));
	checkCudaErrors(hipUnbindTexture(oldForcesTex));
	checkCudaErrors(hipUnbindTexture(oldColTex));

	checkCudaErrors(hipUnbindTexture(cellStartTex));
	checkCudaErrors(hipUnbindTexture(cellEndTex));

	checkCudaErrors(hipUnbindTexture(oldDensAdvTex));
	checkCudaErrors(hipUnbindTexture(oldDensCorrTex));
	checkCudaErrors(hipUnbindTexture(oldP_lTex));
	checkCudaErrors(hipUnbindTexture(oldPreviousPTex));
	checkCudaErrors(hipUnbindTexture(oldAiiTex));

	checkCudaErrors(hipUnbindTexture(oldVelAdvTex));
	checkCudaErrors(hipUnbindTexture(oldForcesAdvTex));
	checkCudaErrors(hipUnbindTexture(oldForcesPTex));
	checkCudaErrors(hipUnbindTexture(oldDiiFluidTex));
	checkCudaErrors(hipUnbindTexture(oldDiiBoundaryTex));
	checkCudaErrors(hipUnbindTexture(oldSumDijTex));
	checkCudaErrors(hipUnbindTexture(oldNormalTex));
#endif

}

//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void pressureSolve(SReal* sortedPos, SReal* sortedVel, SReal* sortedDens, SReal* sortedPres, SReal* sortedForces, SReal* sortedCol, SUint* cellStart, SUint* cellEnd, SUint* gridParticleIndex,
					  SReal* sortedBoundaryPos, SReal* sortedBoundaryVbi,
					  SUint* cellBoundaryStart, SUint* cellBoundaryEnd, SUint* gridBoundaryIndex, SReal* sortedDensAdv, SReal* sortedDensCorr, SReal* sortedP_l,  SReal* sortedPreviousP, 
					  SReal* sortedAii, SReal* sortedVelAdv, SReal* sortedForcesAdv, SReal* sortedForcesP, SReal* sortedDiiFluid, SReal* sortedDiiBoundary, SReal* sortedSumDij, SReal* sortedNormal,
					  SUint numParticles, SUint numBoundaries, SUint numCells)
{
#if USE_TEX
	checkCudaErrors(hipBindTexture(0, oldPosTex, sortedPos, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldVelTex, sortedVel, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDensTex, sortedDens, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPresTex, sortedPres, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldForcesTex, sortedForces, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldColTex, sortedCol, numParticles*sizeof(SVec4)));

	checkCudaErrors(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(SUint)));
	checkCudaErrors(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(SUint)));

	checkCudaErrors(hipBindTexture(0, oldDensAdvTex, sortedDensAdv, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldDensCorrTex, sortedDensCorr, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldP_lTex, sortedP_l, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldPreviousPTex, sortedPreviousP, numParticles*sizeof(SReal)));
	checkCudaErrors(hipBindTexture(0, oldAiiTex, sortedAii, numParticles*sizeof(SReal)));

	checkCudaErrors(hipBindTexture(0, oldVelAdvTex, sortedVelAdv, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldForcesAdvTex, sortedForcesAdv, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldForcesPTex, sortedForcesP, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDiiFluidTex, sortedDiiFluid, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldDiiBoundaryTex, sortedDiiBoundary, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldSumDijTex, sortedSumDij, numParticles*sizeof(SVec4)));
	checkCudaErrors(hipBindTexture(0, oldNormalTex, sortedNormal, numParticles*sizeof(SVec4)));
#endif
	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);

	SUint l=0; 
	SReal rho_avg = 0.f;
	const SReal rd = 1000.f;
	const SReal max_rho_err = 1.f;

	while( ((rho_avg - rd) > max_rho_err) || (l<2))
	{
		//compute sumdijpj
		computeSumDijPj<<<numBlocks, numThreads>>>(
				(SVec4                      *) sortedPos,
				(SVec4                      *) sortedVel,
				sortedDens,
				sortedPres,
				(SVec4                      *) sortedForces,
				(SVec4                      *) sortedCol,
				cellStart,
				cellEnd,
				gridParticleIndex,
				(SVec4					    *)sortedBoundaryPos,
				sortedBoundaryVbi,
				cellBoundaryStart,
				cellBoundaryEnd,
				gridBoundaryIndex,
				sortedDensAdv,
				sortedDensCorr,
				sortedP_l,
				sortedPreviousP,
				sortedAii,
				(SVec4                      *) sortedVelAdv,
				(SVec4                      *) sortedForcesAdv,
				(SVec4                      *) sortedForcesP,
				(SVec4                      *) sortedDiiFluid,
				(SVec4                      *) sortedDiiBoundary,
				(SVec4                      *) sortedSumDij,
				(SVec4                      *) sortedNormal,
				numParticles,
				numBoundaries,
				numCells
		);

		/*SReal maxd =  maxDensity(sortedDensCorr, numParticles);*/
		/*printf("maxd = %f\n", maxd);*/


		/*hipDeviceSynchronize();*/
		//compute pressure
		computePressure<<<numBlocks, numThreads>>>(
				(SVec4                      *) sortedPos,
				(SVec4                      *) sortedVel,
				sortedDens,
				sortedPres,
				(SVec4                      *) sortedForces,
				(SVec4                      *) sortedCol,
				cellStart,
				cellEnd,
				gridParticleIndex,
				(SVec4					    *)sortedBoundaryPos,
				sortedBoundaryVbi,
				cellBoundaryStart,
				cellBoundaryEnd,
				gridBoundaryIndex,
				sortedDensAdv,
				sortedDensCorr,
				sortedP_l,
				sortedPreviousP,
				sortedAii,
				(SVec4                      *) sortedVelAdv,
				(SVec4                      *) sortedForcesAdv,
				(SVec4                      *) sortedForcesP,
				(SVec4                      *) sortedDiiFluid,
				(SVec4                      *) sortedDiiBoundary,
				(SVec4                      *) sortedSumDij,
				(SVec4                      *) sortedNormal,
				numParticles,
				numBoundaries,
				numCells
		);

		/*hipDeviceSynchronize();*/

		//reduce rho_error buffers
		rho_avg = 0.f;
		rho_avg = thrust::reduce(thrust::device_ptr<SReal>(sortedDensCorr),thrust::device_ptr<SReal>(sortedDensCorr+numParticles));
		rho_avg /= numParticles;


		l++;
	}

	/*printf("l = %d\n", l );*/

	computePressureForce<<<numBlocks, numThreads>>>(
				(SVec4                      *) sortedPos,
				(SVec4                      *) sortedVel,
				sortedDens,
				sortedPres,
				(SVec4                      *) sortedForces,
				(SVec4                      *) sortedCol,
				cellStart,
				cellEnd,
				gridParticleIndex,
				(SVec4					    *)sortedBoundaryPos,
				sortedBoundaryVbi,
				cellBoundaryStart,
				cellBoundaryEnd,
				gridBoundaryIndex,
				sortedDensAdv,
				sortedDensCorr,
				sortedP_l,
				sortedPreviousP,
				sortedAii,
				(SVec4                      *) sortedVelAdv,
				(SVec4                      *) sortedForcesAdv,
				(SVec4                      *) sortedForcesP,
				(SVec4                      *) sortedDiiFluid,
				(SVec4                      *) sortedDiiBoundary,
				(SVec4                      *) sortedSumDij,
				(SVec4                      *) sortedNormal,
				numParticles,
				numBoundaries,
				numCells
		);

	/*hipDeviceSynchronize();*/
	iisph_integrate<<<numBlocks, numThreads>>>(
			(SVec4*) sortedPos,
			(SVec4*) sortedVel,
			(SVec4*) sortedVelAdv,
			(SVec4*) sortedForcesP,
			gridParticleIndex,
			numParticles
			);

	/*hipDeviceSynchronize();*/

	/*exit(0);*/

#if USE_TEX
	checkCudaErrors(hipUnbindTexture(oldPosTex));
	checkCudaErrors(hipUnbindTexture(oldVelTex));
	checkCudaErrors(hipUnbindTexture(oldDensTex));
	checkCudaErrors(hipUnbindTexture(oldPresTex));
	checkCudaErrors(hipUnbindTexture(oldForcesTex));
	checkCudaErrors(hipUnbindTexture(oldColTex));

	checkCudaErrors(hipUnbindTexture(cellStartTex));
	checkCudaErrors(hipUnbindTexture(cellEndTex));

	checkCudaErrors(hipUnbindTexture(oldDensAdvTex));
	checkCudaErrors(hipUnbindTexture(oldDensCorrTex));
	checkCudaErrors(hipUnbindTexture(oldP_lTex));
	checkCudaErrors(hipUnbindTexture(oldPreviousPTex));
	checkCudaErrors(hipUnbindTexture(oldAiiTex));

	checkCudaErrors(hipUnbindTexture(oldVelAdvTex));
	checkCudaErrors(hipUnbindTexture(oldForcesAdvTex));
	checkCudaErrors(hipUnbindTexture(oldForcesPTex));
	checkCudaErrors(hipUnbindTexture(oldDiiFluidTex));
	checkCudaErrors(hipUnbindTexture(oldDiiBoundaryTex));
	checkCudaErrors(hipUnbindTexture(oldSumDijTex));
	checkCudaErrors(hipUnbindTexture(oldNormalTex));
#endif

}
/************
*  PCISPH  *
************/
//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void pcisph_internalForces(SReal* sortedPos, SReal* sortedVel, SReal* sortedDens, SReal* sortedPres, SReal* sortedForces, SReal* sortedCol, SUint* cellStart, SUint* cellEnd, SUint* gridParticleIndex,
				SReal* sortedBoundaryPos, SReal* sortedBoundaryVbi, SUint* cellBoundaryStart, SUint* cellBoundaryEnd, SUint* gridBoundaryIndex, SReal* sortedRhoAdv, SReal* sortedPosAdv, SReal* sortedVelAdv, 
				SReal* sortedForcesAdv, SReal* sortedForcesP, SReal* sortedNormal, SUint numParticles, SUint numBoundaries, SUint numCells)
{
#if USE_TEX
	//add texture management someday
#endif

	// thread per particle
	SUint numThreads, numBlocks;
	computeGridSize(numParticles, 64, numBlocks, numThreads);

	computeDensityPressure<<<numBlocks, numThreads>>>(
			(SVec4 *)sortedPos,
			(SVec4 *)sortedVel,
			(SReal *)sortedDens,
			(SReal *)sortedPres,
			(SVec4 *)sortedForces,
			(SVec4 *)sortedCol,
			(SVec4 *)sortedBoundaryPos,
			(SReal *)sortedBoundaryVbi,
			gridParticleIndex,    // input: sorted particle indices
			cellStart,
			cellEnd,
			gridBoundaryIndex,
			cellBoundaryStart,
			cellBoundaryEnd,
			numParticles
	);


#if USE_TEX
	//add texture management someday
#endif
}
//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
void pcisph_pressureSolve(SReal* sortedPos, SReal* sortedVel, SReal* sortedDens, SReal* sortedPres, SReal* sortedForces, SReal* sortedCol, SUint* cellStart, SUint* cellEnd, SUint* gridParticleIndex,
				SReal* sortedBoundaryPos, SReal* sortedBoundaryVbi, SUint* cellBoundaryStart, SUint* cellBoundaryEnd, SUint* gridBoundaryIndex, SReal* sortedRhoAdv, SReal* sortedPosAdv, SReal* sortedVelAdv, 
				SReal* sortedForcesAdv, SReal* sortedForcesP, SReal* sortedNormal, SUint numParticles, SUint numBoundaries, SUint numCells)
{
#if USE_TEX
	//add texture management someday
#endif
	/*printf("pcisph pressure solve\n");*/
}
//==================================================================================================== 
//==================================================================================================== 
//==================================================================================================== 
EXTERN_C_END
